#include "hip/hip_runtime.h"
#include <SFML/Graphics.hpp>
#include <iostream>
#include <cmath>
#include <string>

// Inspired by Javidx9 Mandelbrot fractal for AVX intrinsics - see youtube channel for excellent tutorials

// Only tested on Linux Mint/Ubuntu with GTX 1080 / RTX 2080 cards, CUDA version 11.8
// nvcc cudaMandelbrot.cu -o mandelbrot -O3 -lsfml-graphics -lsfml-window -lsfml-system

int *fractalMatrix = nullptr;
int *CUDAFractal = nullptr;
const int fractalSize = 1024;
int maxIter = 128;
float scale = fractalSize;
int N;

// Create a double struct (sf::Vector only for float)
struct F_CO
{
  double x, y;
};

// Convert pixels to mathematical coordinates (complex numbers)
F_CO toFractal(sf::Vector2i p, F_CO delta, float s){
    F_CO f;
    f.x = ((double) p.x / s*2.5f) + delta.x;
    f.y = ((double) p.y / s*2.5f) + delta.y;
    return f;
}

// GPU PTX code gives slightly faster performance
__global__ void
fractalCallCUDA(F_CO f_co_tl, F_CO f_co_br, sf::Vector2i p_co_tl,
		    sf::Vector2i p_co_br, int *CUDAFractal, int maxIter)
{
  int n = 0;
  double cr = 0.0;
  double ci = 0.0;
  double zr = 0.0;
  double zi = 0.0;
  double re = 0.0;
  double im = 0.0;
  double c1 = 0.0;
  double x_scale;
  double y_scale;

  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int idy = blockDim.y * blockIdx.y + threadIdx.y;

//x_scale = (frac_br_x - frac_tl_x) / (double(pix_br_x) - double(pix_tl_x));
  re = __uint2double_rn (p_co_br.x);
  im = __uint2double_rn (p_co_tl.x);
  cr = __dsub_rn (f_co_br.x, f_co_tl.x);
  ci = __dsub_rn (re, im);
  x_scale = __ddiv_rn (cr, ci);

//y_scale = (frac_br_y - frac_tl_y) / (double(pix_br_y) - double(pix_tl_y));
  re = __uint2double_rn (p_co_br.y);
  im = __uint2double_rn (p_co_tl.y);
  cr = __dsub_rn (f_co_br.y, f_co_tl.y);
  ci = __dsub_rn (re, im);
  y_scale = __ddiv_rn (cr, ci);

  re = __uint2double_rn (idx);
  im = __uint2double_rn (idy);

  cr = __fma_rn (x_scale, re, f_co_tl.x);	//x_pos+x_scale*double(idx);
  ci = __fma_rn (y_scale, im, f_co_tl.y);	//y_pos+y_scale*double(idy);

loop:
  c1 = __dmul_rn (zr, zr);
  re = __fma_rn (zi, zi, -cr);
  re = __dsub_rn (c1, re);
  im = __dmul_rn (zr, zi);
  im = __fma_rn (im, 2.0, ci);	//dmul_rn(re1,2.0);
  zr = re;
  zi = im;
  c1 = __dmul_rn (zr, zr);
  c1 = __fma_rn (zi, zi, c1);
  n = __sad (n, 0, 1);
  if (n < maxIter && c1 < 4.0)
    goto loop;

  CUDAFractal[idy * fractalSize + idx] = n;
}

void fractalCreateCUDA(F_CO tf_co_tl, F_CO tf_co_br, int maxIter)
{
  sf::Vector2i tp_co_tl, tp_co_br;

  // Grid of block and block of threads structure sensitive to performance issues
  // These setting likely optimal for 1024*1024 resolution on RTX/GTX cards
  
  dim3 blocks(8, 8);
  dim3 grid(128, 128);

  tp_co_tl.x = 0;
  tp_co_tl.y = 0;
  tp_co_br.x = fractalSize;
  tp_co_br.y = fractalSize;
  // Seems to be the fastest way to allocate GPU memory
  hipMalloc(&CUDAFractal, N);
  fractalCallCUDA <<< grid, blocks >>> (tf_co_tl, tf_co_br, tp_co_tl,
					   tp_co_br, CUDAFractal, maxIter);
  hipMemcpy(fractalMatrix, CUDAFractal, N, hipMemcpyDeviceToHost);
}

int main()
{
  F_CO f_co_tl, f_co_br, delta, mouseCoordiantesBeforeZoom, mouseCoordinatesAfterZoom;
  sf::Event event;
  sf::Vector2i mouseCoordinates, pan;

// Fractal coordinates displayed at startup
  f_co_tl.x = -2.0f;
  f_co_tl.y = -1.0f;
  f_co_br.x = 1.0f;
  f_co_br.y = 1.0f;
  delta = { -2.0f, -1.2f };

  sf::RenderWindow window (sf::VideoMode (fractalSize, fractalSize),
			   "CUDA Mendelbrot");
  N = fractalSize * fractalSize * sizeof(int);
  // Allocate aligned memory for potential speedup
  fractalMatrix = (int *)aligned_alloc(4096, N);
  // Not freeing memory since program will run until terminated

  sf::Vertex pixel = sf::Vertex(sf::Vector2f (0, 0), sf::Color::White);
  window.clear();

  while (window.isOpen())
  {

      fractalCreateCUDA(f_co_tl, f_co_br, maxIter);

      while (window.pollEvent(event))
      {

    	    mouseCoordinates = sf::Mouse::getPosition(window);
	        if (event.type == sf::Event::Closed)
	          window.close();

          
          // Paning not optimal, unfortunately
	        if (event.type == sf::Event::MouseButtonPressed)
          {
            if (event.mouseButton.button == sf::Mouse::Left)
            {
              pan = sf::Mouse::getPosition(window);
            }
          }

	        if (event.type == sf::Event::MouseButtonReleased)
          {
            if (event.mouseButton.button == sf::Mouse::Left)
            {
              delta.x -= (mouseCoordinates.x - pan.x) / scale;
              delta.y -= (mouseCoordinates.y - pan.y) / scale;
              pan = mouseCoordinates;
            }
          }

	        mouseCoordiantesBeforeZoom = toFractal(mouseCoordinates, delta, scale);

	        if (event.type == sf::Event::MouseWheelMoved)
	        {
	          if (event.mouseWheel.delta > 0)
		        {
		          if (sf::Keyboard::isKeyPressed(sf::Keyboard::LShift))
		          {
		            maxIter += 64;
		            std::cout << maxIter << std::endl;
		          }
		          else
		            scale *= 1.1f;
		        }
	          if (event.mouseWheel.delta < 0)
		        {
		          if (sf::Keyboard::isKeyPressed(sf::Keyboard::LShift))
		          {
		            maxIter -= 64;
		            std::cout << maxIter << std::endl;
		          }
		          else
		            scale *= 0.9f;
		        }
	        }

	      mouseCoordinatesAfterZoom = toFractal(mouseCoordinates, delta, scale);
	      delta.x += (mouseCoordiantesBeforeZoom.x - mouseCoordinatesAfterZoom.x);
	      delta.y += (mouseCoordiantesBeforeZoom.y - mouseCoordinatesAfterZoom.y);
	      f_co_tl = toFractal(sf::Vector2i(0, 0), delta, scale);	// top-left screen coordinates
	      f_co_br = toFractal(sf::Vector2i(fractalSize, fractalSize), delta, scale);	// bottom-right screen coordinates
	    }

    for (int j = 0; j < fractalSize; j++)
    {
	    for (int i = 0; i < fractalSize; i++)
      {
	      int tempCol;
	      double a = 0.1;
	      tempCol = fractalMatrix[j * fractalSize + i];
	      int	red =	(int) ((1.0 -	(0.5 * sinf (a * (double) tempCol) + 0.5)) * 255);
	      int	green =	(int) ((1.0 -	(0.5 * sinf (a * (double) tempCol + 2.094) + 0.5)) * 255);
	      int	blue =	(int) ((1.0 -	(0.5 * sinf (a * (double) tempCol + 4.188) + 0.5)) * 255);
	      pixel.position = sf::Vector2f((float) i, (float) j);
	      pixel.color = sf::Color(red, green, blue);
	      window.draw(&pixel, 1, sf::Points);
      }
    }
    window.display();
  }
  return 0;
}
